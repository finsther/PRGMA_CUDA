
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 16

void multiply_matrix_in_cpu(int a[N][N], int b[N][N], int c[N][N]) {
  int n,m;

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      int sum = 0;

      for (int k = 0; k < N; k++) {
        m = a[i][k];
        n = b[k][j];
        sum += m * n;
      }

      c[i][j] = sum;
    }
  }
}

 __global__ void multiply_matrix_in_gpu(int *a, int *b, int *c) {
   int k, sum = 0;
   int column = threadIdx.x + blockDim.x * blockIdx.x;
   int row = threadIdx.y + blockDim.y * blockIdx.y;

   if (column < N && row < N) {
     for (k = 0; k < N; k++) {
       sum += a[row * N + k] * b[k * N + column];
     }

     c[(row * N) + column] = sum;
   }
 }

int main() {
  int a[N][N], b[N][N], c[N][N];
  int *dev_a, *dev_b, *dev_c;
  int cont,i,j;

  /* initialize both matrix */
  for (i = 0; i < N; i++) {
    cont = 0;

    for (j = 0; j < N; j++) {
      a[i][j] = cont;
      b[i][j] = cont;

      cont++;
    }
  }

  int size = N * N * sizeof(int);

  /* reserve memory */
  hipMalloc((void **) &dev_a, size);
  hipMalloc((void **) &dev_b, size);
  hipMalloc((void **) &dev_c, size);

  hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

  dim3 dimGrid(1, 1);
  dim3 dimBlock(N, N);

  multiply_matrix_in_gpu<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, size, hipMemcpyDeviceToHost);

  /* free memory */
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  /* print values */
  for (int y = 0; y < N; y++) {
    for (int x = 0; x < N; x++) {
      printf("[%d][%d]=%d ", y, x, c[y][x]);
   }

   printf("\n");
  }

  return 0;
}
